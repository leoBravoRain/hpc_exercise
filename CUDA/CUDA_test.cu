
#include <hip/hip_runtime.h>
#include <stdio.h>

void print_matrix(int *s, int N){

  for(int i = 0; i < N; ++i){

    printf("%i | ", s[i]);

  }

}

void init_matrix(int *m, int val, int N){

  for(int i = 0; i < N; ++i){

    m[i] = val;

  }

}

// kernel for run in GPU
// This is a device code beacuse run in GPU
__global__
void saxpy_CUDA(int *s, int a, int *x, int *y, int N){

  for( int i = 0; i < N; ++i){

    s[i] = a*x[i] + y[i];

  }

}

// This is host code because it runs on CPU
int main(int argc, char *argv[]){

  // printf("%i \n", 10);
  if(argc != 2){

    printf("compile must be: \n ./program_name N \n");
  }

  int N = atoi(argv[1]);

  int a = 2;
  // int *s = new int[N];
  // int *x = new int[N];
  // int *y = new int[N];

  int *s, *x, *y;

  // Allocate Unified Memory
  // It's accesible from GPU and CPU
  hipMallocManaged(&s, N*sizeof(int));
  hipMallocManaged(&x, N*sizeof(int));
  hipMallocManaged(&y, N*sizeof(int));

  // Init matrix
  init_matrix(s, 0, N);
  init_matrix(x, 1, N);
  init_matrix(y, 0, N);

  // printf("%i \n", z);

  // saxpy function
  // s = a * x + y
  // saxpy_CUDA(s, a, x, y, N);

  // Launch saxpy_CUDA to kernel for run in GPU
  saxpy_CUDA<<<1,1>>>(s, a, x, y, N)  ;

  // Wait for GPU to finish before accesing on host
  hipDeviceSynchronize();
  
  print_matrix(s, N);

  // Free memory
  hipFree(s);
  hipFree(x);
  hipFree(y);

  return 0;

}

// #include <iostream>
// #include <math.h>
// #include <typeinfo>

// using namespace std;

// // Kernel function to add the elements of two arrays
// __global__
// void add(int n, float *x, float *y)
// {
//   for (int i = 0; i < n; i++)
//     y[i] = x[i] + y[i];
// }

// int main(void)
// {

//   // int N = 1<<20;
//   int N = 20;

//   // printf("%i \n", N);

//   // float *x, *y;
//   int *x, *y;


//   // Allocate Unified Memory – accessible from CPU or GPU
//   cudaMallocManaged(&x, N*sizeof(float));
//   cudaMallocManaged(&y, N*sizeof(float));

//   // initialize x and y arrays on the host
//   for (int i = 0; i < N; i++) {
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//   }

//   // Run kernel on 1M elements on the GPU
//   add<<<1, 1>>>(N, x, y);

//   // Wait for GPU to finish before accessing on host
//   cudaDeviceSynchronize();

//   // Check for errors (all values should be 3.0f)
//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = fmax(maxError, fabs(y[i]-3.0f));
//   std::cout << "Max error: " << maxError << std::endl;

//   // Free memory
//   cudaFree(x);
//   cudaFree(y);
  
//   return 0;
// }