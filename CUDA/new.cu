
#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel
__global__
void add_vectors(int *c, int *a, int *b, int n){

    // printf("Add vectors function\n");

    // printf("n value: %i \n", n);

    int index = threadIdx.x;

    int stride = blockDim.x;

    // int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // int stride = blockDim.x * gridDim.x;

    // printf("%i \n", index);

    for(int i = index; i < n; i += stride){

        // printf("thead %i working with index: %i \n", index, i);

        c[i] = a[i] + b[i];

    }

}

int main(int argc, char *argv[]){

    // Define
    // int *a;
    // int *b;
    // int n = 1000000;
    int n = atoi(argv[1]);
    int nt = atoi(argv[2]);

    int blocks = n/nt;

    printf("blocks: %i \n", blocks);
    printf("works: %i\n", n);
    printf("threads: %i \n", nt);

    // Define and assign vector to dynamic memory
    int *a = (int *) malloc(sizeof(int)*n);
    int *b = (int *) malloc(sizeof(int)*n);
    int *c = (int *) malloc(sizeof(int)*n);

    // Pointers for device memory
    int *d_a;
    int *d_b;
    int *d_c;

    // Assign in device memory
    hipMalloc((void**)&d_a, sizeof(int) * n);
    hipMalloc((void**)&d_b, sizeof(int) * n);
    hipMalloc((void**)&d_c, sizeof(int) * n);

    // init value
    for(int i = 0; i < n; ++i){

        a[i] = 1;
        b[i] = 2;

    }

    // Transfer memory from host to device
    hipMemcpy(d_a, a, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeof(int) * n, hipMemcpyHostToDevice);

    // CPU call
    // add_vectors(c, a, b, n);

    // n = 10000
    // 256 t: 1,1797 ms
    // 1 t: 1,1688 ms

    // n = 1.000.000
    // 1 t: 124,12 ms
    // 256 t: 125,34 ms

    // GPU call: Launch kernel
    add_vectors<<<blocks,nt>>>(d_c, d_a, d_b, n);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Transfer data back to host memory
    hipMemcpy(c, d_c, sizeof(int) * n, hipMemcpyDeviceToHost);

    // for(int i = 0; i < n; ++i){

    //     printf("%i \n", c[i]);

    // }

    hipFree(a);
    hipFree(b);
    hipFree(c);

    free(a);
    free(b);
    free(c);

    return 0;

}

// #include <stdio.h>
// #include <stdlib.h>
// #include <math.h>
// #include <assert.h>
// // #include <cuda.h>
// // #include <cuda_runtime.h>

// #define N 10000000
// #define MAX_ERR 1e-6

// __global__ 
// void vector_add(float *out, float *a, float *b, int n) {

//     printf("running in GPU \n" );

//     for(int i = 0; i < n; i ++){
//         out[i] = a[i] + b[i];
//     }
// }

// int main(){
//     float *a, *b, *out;
//     float *d_a, *d_b, *d_out; 

//     // Allocate host memory
//     a   = (float*)malloc(sizeof(float) * N);
//     b   = (float*)malloc(sizeof(float) * N);
//     out = (float*)malloc(sizeof(float) * N);

//     // Initialize host arrays
//     for(int i = 0; i < N; i++){
//         a[i] = 1.0f;
//         b[i] = 2.0f;
//     }

//     // Allocate device memory
//     cudaMalloc((void**)&d_a, sizeof(float) * N);
//     cudaMalloc((void**)&d_b, sizeof(float) * N);
//     cudaMalloc((void**)&d_out, sizeof(float) * N);

//     // Transfer data from host to device memory
//     cudaMemcpy(d_a, a, sizeof(float) * N, cudaMemcpyHostToDevice);
//     cudaMemcpy(d_b, b, sizeof(float) * N, cudaMemcpyHostToDevice);

//     // Executing kernel 
//     vector_add<<<1,10>>>(d_out, d_a, d_b, N);
    
//     // Transfer data back to host memory
//     cudaMemcpy(out, d_out, sizeof(float) * N, cudaMemcpyDeviceToHost);

//     // Verification
//     for(int i = 0; i < N; i++){
//         assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
//     }
//     printf("out[0] = %f\n", out[0]);
//     printf("PASSED\n");

//     // Deallocate device memory
//     cudaFree(d_a);
//     cudaFree(d_b);
//     cudaFree(d_out);

//     // Deallocate host memory
//     free(a); 
//     free(b); 
//     free(out);
// }